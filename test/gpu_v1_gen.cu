// Generating answers for big test cases
// Using the GPU to generate for certain parts
#include "gpu_utils.h"
#include "gpu_v1.h"
#include "gpu_v2.h"
#include "host.h"
#include "host_utils.h"

#include <iostream>
#include <string>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

using std::cout;
using std::string;

/*
   Output result for each steps
   */

void test_v1_seam(string in_path, bool write_to_file = false) {
  int width, height, channels;

  cout << "Reading from " << in_path << '\n';

  unsigned char *img =
      stbi_load(in_path.c_str(), &width, &height, &channels, 3);

  assert(channels == 3);

  GpuTimer timer;

  cout << "Channels " << channels << " width " << width << " height " << height
       << '\n';

  timer.Start();

  unsigned char *d_in;

  CHECK(hipMalloc(&d_in, sizeof(unsigned char) * 3 * height * width));

  CHECK(hipMemcpy(d_in, img, sizeof(unsigned char) * 3 * height * width,
                   hipMemcpyHostToDevice));

  int *d_gray;
  CHECK(hipMalloc(&d_gray, sizeof(int) * height * width));

  dim3 block_size(1024);
  dim3 grid_size((height * width - 1) / block_size.x + 1);
  V2_grayscale_kernel<<<grid_size, block_size>>>(d_in, height * width, d_gray);

  cout << "Channels " << channels << " width " << width << " height " << height
       << '\n';

  CHECK(hipDeviceSynchronize());
  CHECK(hipGetLastError());

  int *gray = new int[height * width];

  CHECK(hipMemcpy(gray, d_gray, sizeof(int) * height * width,
                   hipMemcpyDeviceToHost));

  if (write_to_file) {
    string out_path = add_ext(in_path, "gray_v1");
    unsigned char *ugray = to_uchar(gray, height * width);
    stbi_write_png(out_path.c_str(), width, height, 1, ugray, width * 1);
    delete[] ugray;
  }

  int *emap = new int[height * width];

  // TODO: replace conv kernel here
  // host_sobel_conv(gray, height, width, emap);

  V1_conv(gray, height, width, emap);

  int *seam = new int[height];
  V1_seam(emap, height, width, seam);

  timer.Stop();

  host_highlight_seam(img, height, width, seam);

  
  string out_path = add_ext(in_path, "seam_v1");

  stbi_write_png(out_path.c_str(), width, height, 3, img, width * 3);

  cout << "Complete in " << timer.Elapsed() << '\n';

}

int main(int argc, char **argv) {
  if (argc < 2)
    return 0;
  printDeviceInfo();

  string file_path(argv[1]);

  // grayscale(file_path);

  test_v1_seam(file_path, true);
}
