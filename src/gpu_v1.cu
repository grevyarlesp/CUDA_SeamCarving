#include "hip/hip_runtime.h"
#include "gpu_utils.h"
#include "gpu_v1.h"
#include <algorithm>
#include <iostream>

using std::cerr;

const int SOBEL_X[] = {
    1, 0, -1, 2, 0, -2, 1, 0, -1,
};

const int SOBEL_Y[] = {1, 2, 1, 0, 0, 0, -1, -2, -1};

__constant__ int kern[9];

__global__ void V1_conv_kernel(int *in, int w, int h, int *out) {

  const int di = 1;
  const int filterWidth = 3;
  int blkR = blockIdx.y * blockDim.y;
  int blkC = blockIdx.x * blockDim.x;
  int r = blkR + threadIdx.y;
  int c = blkC + threadIdx.x;
  extern __shared__ int s_in[];
  s_in[threadIdx.y * blockDim.x + threadIdx.x] = in[r * w + c];
  __syncthreads();
  if (threadIdx.x < blockDim.x && threadIdx.y < blockDim.y) {
    int ind = r * w + c;
    int sum = 0;
    for (int i = 0; i < filterWidth; i++)
      for (int j = 0; j < filterWidth; j++) {
        int ki = i * filterWidth + j;
        int x = threadIdx.x - di + j;
        int y = threadIdx.y - di + i;
        if (blkC + x < 0 || blkC + x >= w)
          x = (blkC + x < 0) ? 0 : w - 1 - blkC;
        if (blkR + y < 0 || blkR + y >= h)
          y = (blkR + y < 0) ? 0 : h - 1 - blkR;
        if (x < blockDim.x && y < blockDim.y) {
          int convind = y * blockDim.x + x;
          sum += kern[ki] * s_in[convind];
        } else {
          y += blkR;
          x += blkC;
          int convind = y * w + x;
          sum += kern[ki] * in[convind];
        }
      }
    out[ind] = sum;
  }
}

// __global__ void min_kern(int* in, int n, int* out)
// {
//   int numElemsBeforeBlk = blockIdx.x * blockDim.x * 2;
//     for (int stride = blockDim.x; stride >= 1; stride /= 2)
//     {
//         int i = numElemsBeforeBlk + threadIdx.x;
//         if (threadIdx.x < stride)
//             if ((i + stride < n) && (in[i] != in[i + stride]))
//                 in[i] = in[i] < in[i + stride] ? in[i] : i[i + stride];

//         __syncthreads();
//     }
//     if (threadIdx.x == 0)
//         out[blockIdx.x] = in[numElemsBeforeBlk];
// }

#define TILE_DIM 32
#define BLOCK_ROWS 8

__global__ void Tpose_kern(int *d_in, int height, int width, int *out) {
  __shared__ int tile[TILE_DIM][TILE_DIM];
  int i_n = blockIdx.x * TILE_DIM + threadIdx.x;
  int i_m = blockIdx.y * TILE_DIM + threadIdx.y;

  int i;
  for (i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    if (i_n < height && (i_m + i) < width) {
      tile[threadIdx.y + i][threadIdx.x] = d_in[(i_m + i) * width + i_n];
    }
  }
  __syncthreads();

  i_n = blockIdx.y * TILE_DIM + threadIdx.x;
  i_m = blockIdx.x * TILE_DIM + threadIdx.y;

  for (i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    if (i_n < height && (i_m + i) < width) {
      out[(i_m + i) * width + i_n] = tile[threadIdx.x][threadIdx.y + i];
    }
  }
}

__global__ void V1_grayscale_kernel(unsigned char *d_in, int height, int width,
                                    int *out) {

  int r = blockDim.y * blockIdx.y + threadIdx.y;
  int c = blockDim.x * blockIdx.x + threadIdx.x;

  if (r >= height || c >= width)
    return;
  int pos = (r * width + c) * 3;
  int ans = (d_in[pos] + d_in[pos + 1] + d_in[pos + 2]) / 3;
  out[pos] = ans;
}

void V1_grayscale(unsigned char *in, int height, int width, int *out, int block_size) {
  unsigned char *d_in;
  int *d_out;
  hipMalloc(&d_in, height * width * sizeof(unsigned char));
  hipMalloc(&d_out, height * width * sizeof(int));
  hipMemcpy(d_in, in, height * width * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  dim3 blockSize(block_size, block_size);
  dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
  V1_grayscale_kernel<<<gridSize, blockSize>>>(d_in, height, width, d_out);
  hipMemcpy(out, d_out, height * width * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
}

void V1_conv(int *in, int height, int width, bool sobelx, int *out) {
  int *d_in, *d_out;
  size_t imgSize = width * height * sizeof(int);
  size_t kernSize = 9 * sizeof(int);
  CHECK(hipMalloc(&d_in, imgSize));
  CHECK(hipMalloc(&d_out, imgSize));
  CHECK(hipMemcpy(d_in, in, imgSize, hipMemcpyHostToDevice));
  CHECK(hipMemcpyToSymbol(HIP_SYMBOL(kern), SOBEL_X, kernSize));
  CHECK(hipMemcpyToSymbol(HIP_SYMBOL(kern), SOBEL_Y, kernSize));

  dim3 blockSize(32, 32);
  dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
  V1_conv_kernel<<<gridSize, blockSize, width * height * sizeof(int)>>>(d_in, width, height,
                                                               d_out);
  hipDeviceSynchronize();
  hipGetLastError();
  hipMemcpy(out, d_out, width * height * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_in);
  CHECK(hipFree(d_out));
}

/*
   Dynamic programming kernel for finding seam
   */
__global__ void V1_dp_kernel(int *d_in, int *d_dp, int *d_trace, int width,
                             int row) {

  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (col >= width)
    return;

  if (row == 0) {
    d_dp[col] = d_in[col];
    return;
  }

  int ans = -1;
  int tr = -1;

  for (int j = -1; j <= 1; ++j) {
    int col_ = col + j;
    if (col_ < 0 || col_ >= width)
      continue;

    int tmp = d_dp[(row - 1) * width + col_];

    if (ans == -1 || tmp < ans) {
      ans = tmp;
      tr = col_;
    }
  }

  d_trace[row * width + col] = tr;

#ifdef V1_DEBUG
  printf("%d %d %d\n", row, col, d_in[row * width + col]);
#endif

  d_dp[row * width + col] = ans + d_in[row * width + col];
#ifdef V1_DEBUG
  printf("DP %d %d %d\n", row, col, d_dp[row * width + col]);
#endif
}

/*
Input: n * m energy map
Output: result + time
*/
double V1_seam(int *in, int height, int width, int *out, int blocksize) {

#ifdef V1_DEBUG
  cerr << "==================================\n";
  cerr << "Debug for V1_seam" << '\n';
  cerr << "==================================\n";
#endif

  GpuTimer timer;
  timer.Start();

  dim3 grid_size((width - 1) / blocksize + 1);
  dim3 block_size(blocksize);

  int *d_in;
  CHECK(hipMalloc(&d_in, height * width * sizeof(int)));
  CHECK(hipMemcpy(d_in, in, height * width * sizeof(int),
                   hipMemcpyHostToDevice));

  int *d_dp;
  CHECK(hipMalloc(&d_dp, height * width * sizeof(int)));

  int *d_trace;
  CHECK(hipMalloc(&d_trace, height * width * sizeof(int)));

  for (int i = 0; i < height; ++i) {
#ifdef V1_DEBUG_ROW
    cerr << i << ' ';
#endif
    V1_dp_kernel<<<grid_size, block_size>>>(d_in, d_dp, d_trace, width, i);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
  }

#ifdef V1_DEBUG_ROW
    cerr << i << '\n';
#endif

  // trace back
  int *trace = new int[height * width];

  CHECK(hipMemcpy(trace, d_trace, height * width * sizeof(int),
                   hipMemcpyDeviceToHost));

  int *dp = new int[width];
  CHECK(hipMemcpy(dp, d_dp + (height - 1) * width, width * sizeof(int),
                   hipMemcpyDeviceToHost));

  // fix trace
  int pos = (int)(std::min_element(dp, dp + width) - dp);

#ifdef V1_DEBUG
  cerr << "Pos = " << pos << '\n';
#endif

  for (int i = height - 1; i >= 0; --i) {
    out[i] = pos;

    if (i > 0)
      pos = trace[i * width + pos];
  }

  delete[] trace;
  delete[] dp;
  CHECK(hipFree(d_in));
  CHECK(hipFree(d_dp));
  CHECK(hipFree(d_trace));

  timer.Stop();

#ifdef DEBUG
  cerr << "End of debug for V1_seam" << '\n';
  cerr << "==================================\n";
#endif

  return timer.Elapsed();
}

__global__ void V1_seam_removal_kernel() {}

__global__ void V1_seam_add_kernel() {}

void v1_in_to_seam(unsigned char *in, int height, int width, char *out,
                   int blocksize) {

  unsigned char *d_in;
  CHECK(hipMalloc(&d_in, sizeof(unsigned char) * 3 * height * width));

  int *d_gray;
  CHECK(hipMalloc(&d_gray, sizeof(char) * height * width));

  dim3 block_size(blocksize, blocksize);
  dim3 grid_size((height - 1) / blocksize + 1, (width - 1) / blocksize + 1);

  V1_grayscale_kernel<<<grid_size, block_size>>>(d_in, height, width, d_gray);
}
