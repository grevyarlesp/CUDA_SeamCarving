#include "hip/hip_runtime.h"
#include "gpu_utils.h"
#include "gpu_v1.h"
#include "gpu_v2.h"
#include <__clang_cuda_builtin_vars.h>
#include <algorithm>
#include <iostream>

using std::cerr;

__global__ void V2_grayscale_kernel(unsigned char *d_in, int num_pixels,
                                    int *out) {

  int pos = blockDim.x * blockIdx.x + threadIdx.x;
  if (pos >= num_pixels) return;

  int pos_ = pos * 3;
  int ans = (d_in[pos_] * 3 + d_in[pos_ + 1] * 6 + d_in[pos_ + 2]) / 10;

  out[pos] = ans;
}


/*
   Manual merge, Parallelized  DP.
   Split into 4 parts
   */

__global__ void V2_conv_kernel(int *d_in, int height, int width, int *d_out) {

}

__device__ int bCount;
__device__ int done;

__global__ void V2_dp_kernel(int *d_in, int height, int width, int *d_out, int *d_dp) {
  __shared__ int bi;

  if (threadIdx.x == 0) {
    bi = atomicAdd(&bCount, 1);
  }

  __syncthreads();

  int tidx = threadIdx.x;
  

  int cnt = gridDim.x * blockDim.x;
  int row = bi / cnt;
  int col = bi % cnt;

  if (row >= height || col >= width) {
    return;
  }

  // first row of the block
  if (threadIdx.y == 0) {
    
    
  }

  // wait for the required number of threads to complete 

  // merge results.


}

// tracing so we don't have to copy
__global__ void trace_kernel(int *d_trace) {
  
}



// overlapping convolution
double V2_dp_seam() {
  GpuTimer timer;
  timer.Start();




  timer.Stop();
  return timer.Elapsed();
}
