#include "hip/hip_runtime.h"
#include "gpu_utils.h"
#include "gpu_v1.h"
#include "gpu_v1_2.h"
#include <algorithm>
#include <iostream>

using std::cerr;

/*
   Dynamic programming kernel for finding seam
   */
__global__ void V1_3_dp_kernel(int *d_in, int *d_dp_prev, int *d_dp_cur,
                               int *d_trace, int height, int width, int row, int *d_out) {

  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (col >= width)
    return;

  if (row == 0) {
    d_dp_cur[col] = d_in[col];
    return;
  }

  int pos = row * width + col;

  // middle
  int ans = d_dp_prev[col];
  int tr = col;

  for (int j = -1; j <= 1; j += 2) {
    int col_ = col + j;
    if (col_ < 0 || col_ >= width)
      continue;

    int tmp = d_dp_prev[col_];

    if (tmp < ans) {
      ans = tmp;
      tr = col_;
    }
  }

  d_trace[pos] = tr;
  d_dp_cur[col] = ans + d_in[pos];

  if (row == height - 1) {

    int pos = col;

    for (int i = height - 1; i >= 0; --i) {
      d_out[col * height + i] = pos;
      if (i > 0) {
        pos = d_trace[i * width + pos];
      }
    }
  }
}

int completed_trace = 0;


/*
Input: n * m energy map
Output: result + time
*/
double V1_3_seam(int *in, int height, int width, int *out, int blocksize) {

  GpuTimer timer;
  timer.Start();

  dim3 grid_size((width - 1) / blocksize + 1);
  dim3 block_size(blocksize);

  int matBytes = height * width * sizeof(int);

  int *d_in;
  CHECK(hipMalloc(&d_in, matBytes));
  CHECK(hipMemcpy(d_in, in, matBytes, hipMemcpyHostToDevice));

  // int *d_dp;
  // CHECK(hipMalloc(&d_dp, matBytes));

  int row_sz = width * sizeof(int);

  int *d_dp_cur;
  CHECK(hipMalloc(&d_dp_cur, row_sz));

  int *d_dp_prev;
  CHECK(hipMalloc(&d_dp_prev, row_sz));

  int *d_trace;
  CHECK(hipMalloc(&d_trace, matBytes));

  int *d_out;
  CHECK(hipMalloc(&d_out, matBytes));

  int *trace = new int[height * width];

  // CHECK(hipHostRegister(in, matBytes, hipHostRegisterDefault));
  // CHECK(hipHostRegister(trace, matBytes, hipHostRegisterDefault));

  // hipStream_t *strems;
  // streams = (hipStream_t *)malloc(sizeof(hipStream_t) * nStreams);

  for (int i = 0, start = 0; i < height; ++i, start+=width) {
    V1_3_dp_kernel<<<grid_size, block_size>>>(d_in, d_dp_prev, d_dp_cur,
                                              d_trace, height, width, i, d_out);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    int *tmp = d_dp_prev;
    d_dp_prev = d_dp_cur;
    d_dp_cur = tmp;
  }

  CHECK(hipMemcpy(trace, d_trace, matBytes, hipMemcpyDeviceToHost));

  int *dp = new int[width];
  CHECK(hipMemcpy(dp, d_dp_prev, row_sz, hipMemcpyDeviceToHost));

  int pos = (int)(std::min_element(dp, dp + width) - dp);

#if V1_3_DEBUG
  std::cout << "Tracing\n";
#endif

  CHECK(hipMemcpy(out, d_out + pos * height, height * sizeof(int), hipMemcpyDeviceToHost));
  timer.Stop();

  delete[] trace;
  delete[] dp;
  CHECK(hipFree(d_in));
  CHECK(hipFree(d_dp_cur));
  CHECK(hipFree(d_out));
  CHECK(hipFree(d_dp_prev));
  CHECK(hipFree(d_trace));
  return timer.Elapsed();
}


